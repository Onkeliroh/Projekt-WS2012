//compile with "nvcc -arch=sm_20 -lcudart"

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add (int *a, int *b, int *c)
{
	*c = *a + *b;
}

int main( void )
{
	int a,b,c;
	int *a2,*b2,*c2;

	hipMalloc((void**) &a2, sizeof(int));
	hipMalloc((void**) &b2, sizeof(int));
	hipMalloc((void**) &c2, sizeof(int));

	a = 5;
	b = 42;

	hipMemcpy(a2, &a , sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(b2, &b , sizeof(int), hipMemcpyHostToDevice);

	add<<<1,1>>>(a2,b2,c2);

	hipMemcpy(&c, c2, sizeof(int), hipMemcpyDeviceToHost);
	
	hipFree(a2);
	hipFree(b2);
	hipFree(c2);

	printf("%i\n",a);
	printf("%i\n",b);	
	printf("%i\n",c);
}
