#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void add (int *a, int *b, int *c)
{
	*c=*a+*b;
}

int main( void )
{
	int a,b,c;
	int *a2,*b2,*c2;

	hipMalloc((void**) &a2, sizeof(int));
	hipMalloc((void**) &b2, sizeof(int));
	hipMalloc((void**) &c2, sizeof(int));

	a = 5;
	b = 42;

	hipMemcpy(a2, &a , sizeof(int), cudaMemcpyHosttoDevice);
	hipMemcpy(b2, &b , sizeof(int), cudaMemcpyHosttoDevice);

	add<<<1,1>>>(a,b,c);

	hipMemcpy(&c, c2, sizeof(int), cudaMemcpyDevicetoHost);
	
	printf("%i\n",c);
}
