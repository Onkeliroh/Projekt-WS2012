#include "hip/hip_runtime.h"
//compile with "nvcc -arch=sm_20 -lcudart"
#include <stdio.h>

#define Blocksize 2


__global__ void add (int *a, int *b, int *c)
{
	__shared__ int* dings;
	__shared__ int dings[1];
	if (threadIdx.x == 0){
		dings[0]=24;
		c[threadIdx.x] = dings[0];
		__syncthreads();
	}
	else{
		c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x] + dings[0];
		__syncthreads();
	}
}

int main( void )
{
	int *a, *b, *c;
	int *a2,*b2,*c2;
	int size = Blocksize * sizeof(int);

	hipMalloc((void**) &a2, size);
	hipMalloc((void**) &b2, size);
	hipMalloc((void**) &c2, size);

	a = (int *)malloc(size); a[0] = 43; a[1] = 42;
	b = (int *)malloc(size); b[0] = 45; b[1] = 44;
	c = (int *)malloc(size);

	hipMemcpy(a2, a , size, hipMemcpyHostToDevice);
	hipMemcpy(b2, b , size, hipMemcpyHostToDevice);

	add<<<1,Blocksize>>>(a2,b2,c2);

	hipMemcpy(c, c2, size, hipMemcpyDeviceToHost);
	
	hipFree(a2);
	hipFree(b2);
	hipFree(c2);

	printf("%i\t%i\n",a[0],a[1]);
	printf("%i\t%i\n",b[0],b[1]);	
	printf("%i\t%i\n",c[0],c[1]);
}
