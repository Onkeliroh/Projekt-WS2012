//compile with:

#include <hip/hip_runtime.h>
#include <stdio.h>

#define Blocksize 10

__global__ void compute( char*, char*);
__device__ __host__ void algorithm(char*, char*);

__host__
int main (void)
{
	char* targets;
	char* targets2;
	char* result;
	char* result2;

	int size = Blocksize * sizeof(char);

	//speicherreservieren
	hipMalloc((void **) &targets2, size);
	hipMalloc((void **) &result2, size);

	targets	= (char *) malloc(size);
	result	= (char *) malloc(size); 

	targets[0] = 'A';
	targets[1] = 'B';
	targets[2] = 'C';
	targets[3] = 'D';
	targets[4] = 'E';
	targets[5] = 'F';
	targets[6] = 'G';
	targets[7] = 'H';
	targets[8] = 'I';
	targets[9] = 'J';


	hipMemcpy(targets2, targets, size, hipMemcpyHostToDevice);
	hipMemcpy(result2, result, size, hipMemcpyHostToDevice);

	compute<<<1,Blocksize>>>(targets2,result2);

	hipMemcpy(result, result2, size, hipMemcpyDeviceToHost);
	hipMemcpy(targets, targets2, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < Blocksize; ++i)
	{
		printf("%c\t",targets[i]);
		printf("%c\n",result[i]);
	}
	hipFree(targets2);
	hipFree(result2);
	return 0;
}

__global__
void compute(char* target, char* result){
	__shared__ char solution[1];
	if (threadIdx.x == 0){
		solution[0] = 'D';
		result[threadIdx.x] = '-';
		__syncthreads();
	}
	else{
		if (target[threadIdx.x] == solution[0]){
			result[threadIdx.x] = '+';
		}
		else{
			result[threadIdx.x] = '-';
		}
		__syncthreads();
	}
}


__device__ __host__ 
void algorithm(char* input, char* result){
	//vorläufig
	result[0] = result[0];
}
