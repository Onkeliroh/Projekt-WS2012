//compile with: nvcc -arch=sm_20 -lcudart

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

#define Blocksize 10

__global__ void compute(char*, char*);
__device__ __host__ void algorithm(char*, char*);
__device__ int cudaMemCmp(const char*, const char*, int);

__host__
int main (void)
{
	char	targets[100];
	char* targets2;
	char  result[100];
	char* result2;

	int size = Blocksize * sizeof(char);

  memset(targets, 0, 100);
	memset(result, 0, 100);

	//speicherreservieren
	hipMalloc( &targets2, (size * 10));
	hipMalloc( &result2, (size * 10));


	strcpy(targets + 0 * 10, "Baum"); 
	strcpy(targets + 1 * 10, "Haus"); 
	strcpy(targets + 2 * 10, "Daniel"); 
	strcpy(targets + 3 * 10, "Blubber"); 
	strcpy(targets + 4 * 10, "Tackle"); 
	strcpy(targets + 5 * 10, "Pokeball"); 
	strcpy(targets + 6 * 10, "Computer"); 
	strcpy(targets + 7 * 10, "Dingens"); 		
	strcpy(targets + 8 * 10, "Baum"); 
	strcpy(targets + 9 * 10, "Baum"); 


	hipMemcpy(targets2, targets, 100, hipMemcpyHostToDevice);
	hipMemcpy(result2, result, 100, hipMemcpyHostToDevice);

	compute<<<1,Blocksize>>>(targets2,result2);

	hipMemcpy(result, result2, 10 *size, hipMemcpyDeviceToHost);
	hipMemcpy(targets, targets2, 10 * size, hipMemcpyDeviceToHost);

	hipFree(targets2);
	hipFree(result2);

	return 0;
}

__global__
void compute(char* target, char* result){
	__shared__ char solution[10];
	if (threadIdx.x == 0){
		memcpy(solution, "p<82/.99", 8);
		result[Blocksize * threadIdx.x] = '-';
		__syncthreads();
	}
	else{
		algorithm((target + (Blocksize * threadIdx.x)), (result + Blocksize * threadIdx.x));
		if (cudaMemCmp((result + Blocksize * threadIdx.x), solution, Blocksize) == 1){
			printf("Found %s for %s\n",(result + Blocksize * threadIdx.x), (target + Blocksize * threadIdx.x));
		}
		else{
			printf("NOT Found %s for %s\n",(result + Blocksize * threadIdx.x), (target + Blocksize * threadIdx.x));//		*(result + Blocksize * threadIdx.x) = '!';
		}
		__syncthreads();
	}
}

__device__ __host__ 
void algorithm(char* input, char* result){
	for (int i = 0; i < Blocksize; ++i){
		if (((int)input[i] < 127) && ((int)input[i]>32))
			result[i] = input[i] % 83 +32;
		else
			break;
	}
}

__device__
int
cudaMemCmp(const char* left, const char* right, int length)
{
		int result = 1;
		while(result && (length > 0)){
		--length;
		result &= (left[length] == right[length]);
		}
		return result;
}
