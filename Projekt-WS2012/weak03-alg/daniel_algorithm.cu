#include <mack/algorithms/daniel_algorithm.cuh>
#include <hip/hip_runtime.h>
#include <mack/algorithms/daniel_help.cuh>

namespace mack{

	__device__
	daniel_algorithm::daniel_algorithm()
	: _target_size(16)
	{}

	__device__ __host__
	void
	daniel_algorithm::compute_target(mack::core::canidate key_canidate, unsigned char* result) const
	{
		daniel_CTX	context;
		//unsigned char digest [16];

		daniel_init( &context );
		daniel_update ( &context, (void*)(key_canidate.value), key_canidate.length);
		daniel_final( result, &context );
	}

	__device__ __host__
	unsigned int
	daniel_algorithm::get_target_size(size_t length) const
	{
		return _target_size;
	}

	__device__ __host__
	unsigned char*
	daniel_algorithm::get_name() const
	{
		return (unsigned char*)"Daniels Algorithmus";
	}

	__device__ __host__
	void
	daniel_algorithm::init(boost::program_options::variables_map const& options) { }

	__device__ __host__
	daniel_algorithm::~daniel_algorithm() { }

}

