#include <mack/algorithms/daniel_algorithm.cuh>
#include <hip/hip_runtime.h>

namespace mack{

	__device__
	daniel_algorithm::daniel_algorithm()
	: _target_size(16)
	{}

	__device__ __host__
	void
	daniel_algorithm::compute_target(mack::core::canidate key_canidate, unsigned char* result) const
	{
		//hier arbeiten !!! 
		//unsigned char digest [16];
	 	result = static_cast<unsigned int>(key_canidate.
	}

	__device__ __host__
	unsigned int
	daniel_algorithm::get_target_size(size_t length) const
	{
		return _target_size;
	}

	__device__ __host__
	unsigned char*
	daniel_algorithm::get_name() const
	{
		return (unsigned char*)"Daniels Algorithmus";
	}

	__device__ __host__
	daniel_algorithm::~daniel_algorithm() { }
}

